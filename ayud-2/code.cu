
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void add(float *c, float* a, float *b, int values){
	int blockD = blockDim.x;
	int blockX = blockIdx.x;
	int threadX = threadIdx.x;

	int i = blockX * blockD + threadX;
	if(i < values)
		c[i] = a[i] + b[i];
	//printf("Hello Im thread %d in block %d of %d threads\n", threadX, blockX, blockD);
}

__host__ int main (int argc, char *argv[]){

	int numValues = atoi(argv[1]);
	int blocksize = atoi(argv[2]);
	
	printf("Using program with %d values and %d blocksize\n", numValues, blocksize);

	float *c = (float*)malloc(numValues*sizeof(float));
	float *a = (float*)malloc(numValues*sizeof(float));
        float *b = (float*)malloc(numValues*sizeof(float));
	
	float *c_d, *b_d, *a_d;
	hipMalloc((void**)&c_d, numValues*sizeof(float));
	hipMalloc((void**)&b_d, numValues*sizeof(float));
	hipMalloc((void**)&a_d, numValues*sizeof(float));

	for(int i=0; i < numValues; i++){
		c[i] = 0.0;
		a[i] = 3.0;
		b[i] = 5.0;
	}
	printf("Done init\n");
	int numBlocks = numValues/blocksize;
	
	printf("Copying arrays from host to device\n");
	hipMemcpy(a_d, a, numValues*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, numValues*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(c_d, c, numValues*sizeof(float), hipMemcpyHostToDevice);

	add<<<numBlocks,blocksize>>>(c_d, a_d, b_d, numValues);
	hipDeviceSynchronize();
	
	printf("Copying values back to host\n");

	hipMemcpy(c, c_d, numValues*sizeof(float), hipMemcpyDeviceToHost);
	
	for(int i=0; i < numValues; i++)
		printf("C[%d] = %f\n", i, c[i]);
	return 0;
}
