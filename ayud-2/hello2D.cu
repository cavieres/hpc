
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello2D()
{
	int blocksize = blockIdx.y * blockIdx.x;
	int blockId = gridDim.x * blockIdx.y + blockIdx.x;
	int tid = blockId * blocksize + blockDim.x * threadIdx.y + threadIdx.x;
	
	printf("I am thread (%d, %d) in block (%d, %d). Global thread ID = %d\n", threadIdx.y, threadIdx.x, blockIdx.y, blockIdx.x, tid);
}

__host__ int main(void)
{
	dim3 blocksize;
	dim3 gridsize;
	
	// 2D blocks in a 2D grid.
	gridsize.x = 3;
	gridsize.y = 2;
	blocksize.x = 3;
	blocksize.y = 4;
	
	hello2D<<<gridsize, blocksize>>>();
	hipDeviceSynchronize();
	return 0;
}
