
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <getopt.h>
#include <string.h>
#include <omp.h>

// Save wave state as binary file.
int setWaveSpace(int N, char *f, float *waveSpace)
{
	FILE *filestream = fopen(f, "w+b");
	
	if (!filestream)
	{
		printf("Couldn't open file.\n");
		return 1;
	}
	
	for (int i = 0; i < N; i++)
	{
		fwrite(&waveSpace[i], sizeof(float), N * N, filestream);
		
		// Print wave space as CSV format.
		for (int j = 0; j < N; j++)
			printf("%f;", waveSpace[N * i + j]);
		printf("\n");
	}
	
	fclose(filestream);

	return 0;
}

// Read a wave state from binary file.
int getWaveSpace(int N, char *f)
{
	FILE *filestream = fopen(f, "r+b");
	float ws[N * N];

	if (!filestream)
	{
		return 1;
	}

	fread(&ws, sizeof(float), N * N, filestream);
	
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%f;", ws[N * i + j]);
		printf("\n");
	}
	
	fclose(filestream);

	return 0;
}

// Filling wave space with T = 1.
void fillSpaceFirstStep(int N, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1)
{
	for (int i = 1; i < N; i++)
		for (int j = 1; j < N - 1; j++)
			waveSpace[N * i + j] = waveSpaceTMin1[N * i + j] + (c * c)/2 * (dt/dd * dt/dd) * (waveSpaceTMin1[N * (i + 1) + j] + waveSpaceTMin1[N * (i - 1) + j] + waveSpaceTMin1[N * i + (j - 1)] + waveSpaceTMin1[N * i + (j + 1)] - 4 * waveSpaceTMin1[N * i + j]);
}

// Setting wave space as a grid with values "20" in center
// sorrounding with zero values.
void initializeSpace(int N, float *waveSpace)
{
	for (int i = 0; i < N * N; i++)
		waveSpace[i] = 0;
	
	for (int i = 0.4 * N; i < 0.6 * N; i++)
		for (int j = 0.4 * N; j < 0.6 * N; j++)
			waveSpace[N * i + j] = 20;
}

// Filling wave space with T > 1.
__global__ void fillSpaceTSteps(int N, int T, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1, float *waveSpaceTMin2)
{

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
//waveSpace[N * i + j] = 2 * waveSpaceTMin1[N * i + j];
	waveSpace[N * i + j] = 2 * waveSpaceTMin1[N * i + j] - waveSpaceTMin2[N * i + j] + (c * c) * (dt/dd * dt/dd) * (waveSpaceTMin1[N * (i + 1) + j] + waveSpaceTMin1[N * (i - 1) + j] + waveSpaceTMin1[N * i + (j - 1)] + waveSpaceTMin1[N * i + (j + 1)] - 4 * waveSpaceTMin1[N * i + j]);
	__syncthreads();

	if (threadIdx.x == 0)
	{
		memcpy(waveSpaceTMin2, waveSpaceTMin1, N * N * sizeof(float));
		memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
	}
}

__host__ int main(int argc, char **argv)
{
	double start = omp_get_wtime();

	dim3 numblocks;
	dim3 sizeblocks;	
	
	int N, T, X, Y, t;
	
	int selection;
	char *f;
	
	while ((selection = getopt(argc, argv, "N:T:X:Y:f:t:")) != -1)
	{
		switch(selection) {
			case 'N':
				N = atoi(optarg);
				break;
			case 'T':
				T = atoi(optarg);
				break;
			case 'X':
				X = atoi(optarg);
				break;
			case 'Y':
				Y = atoi(optarg);
				break;
			case 'f':
				f = optarg;
				remove(f);
				break;
			case 't':
				t = atoi(optarg);
				break;
		}
	}
	
	if (t > T)
	{
		printf("Error: output iteration can't be bigger than step quantity.\n");
		exit(1);
	}

	numblocks.x = (int)ceil((float)N/X);
	numblocks.y = (int)ceil((float)N/Y);

	sizeblocks.x = X;
	sizeblocks.y = Y;

	// Setting wave spaces, saving states t, t - 1 and t - 2.
	float *waveSpace, *waveSpaceTMin1, *waveSpaceTMin2;
	waveSpace = (float *)malloc(N * N * sizeof(float));
	waveSpaceTMin1 = (float *)malloc(N * N * sizeof(float));
	waveSpaceTMin2 = (float *)malloc(N * N * sizeof(float));
	
	// Setting wave spaces in GPU.
	float *waveSpace_d, *waveSpaceTMin1_d, *waveSpaceTMin2_d;
	hipMalloc((void**)&waveSpace_d, N * N * sizeof(float));
	hipMalloc((void**)&waveSpaceTMin1_d, N * N * sizeof(float));
	hipMalloc((void**)&waveSpaceTMin2_d, N * N * sizeof(float));

	float c = 1.0;
	float dt = 0.1;
	float dd = 2.0;

	for (int step = 0; step <= T; step++)
	{
		switch(step)
		{
			case 0:
				initializeSpace(N, waveSpace);
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
			case 1:
				fillSpaceFirstStep(N, c, dt, dd, waveSpace, waveSpaceTMin1);
				memcpy(waveSpaceTMin2, waveSpaceTMin1, N * N * sizeof(float));
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));

				// Copying values from CPU to GPU.
				hipMemcpy(waveSpace_d, waveSpace, N * N * sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(waveSpaceTMin1_d, waveSpaceTMin1, N * N * sizeof(float), hipMemcpyHostToDevice);
				hipMemcpy(waveSpaceTMin2_d, waveSpaceTMin2, N * N * sizeof(float), hipMemcpyHostToDevice);

				break;
			default:
				

				// Executing kernel from step 2.
				fillSpaceTSteps<<<numblocks,sizeblocks>>>(N, T, c, dt, dd, waveSpace_d, waveSpaceTMin1_d, waveSpaceTMin2_d);
				hipDeviceSynchronize();
				break;
		}
	}

	if (T > 1)
		hipMemcpy(waveSpace, waveSpace_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

	setWaveSpace(N, f, waveSpace);

	double end = omp_get_wtime();
	
	printf("Time spent: %f\n", end - start);
	
	return 0;
}
