
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <getopt.h>
#include <string.h>
#include <omp.h>

// Save wave state as binary file.
int setWaveSpace(int N, char *f, float *waveSpace)
{
	FILE *filestream = fopen(f, "w+b");
	
	if (!filestream)
	{
		printf("Couldn't open file.\n");
		return 1;
	}
	
	for (int i = 0; i < N; i++)
	{
		fwrite(&waveSpace[i], sizeof(float), N * N, filestream);
		// Print wave space as CSV format.
		for (int j = 0; j < N; j++)
			printf("%f;", waveSpace[N * i + j]);
		printf("\n");
	}
	
	fclose(filestream);

	return 0;
}

// Read a wave state from binary file.
int getWaveSpace(int N, char *f)
{
	FILE *filestream = fopen(f, "r+b");
	float ws[N * N];

	if (!filestream)
	{
		return 1;
	}

	fread(&ws, sizeof(float), N * N, filestream);
	
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%f;", ws[N * i + j]);
		printf("\n");
	}
	
	fclose(filestream);

	return 0;
}

// Filling wave space with T = 1.
__device__ void fillSpaceFirstStep(int N, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1)
{
	for (int i = 1; i < N; i++)
		for (int j = 1; j < N - 1; j++)
			waveSpace[N * i + j] = waveSpaceTMin1[N * i + j] + (c * c)/2 * (dt/dd * dt/dd) * (waveSpaceTMin1[N * (i + 1) + j] + waveSpaceTMin1[N * (i - 1) + j] + waveSpaceTMin1[N * i + (j - 1)] + waveSpaceTMin1[N * i + (j + 1)] - 4 * waveSpaceTMin1[N * i + j]);
}

// Setting wave space as a grid with values "20" in center
// sorrounding with zero values.
__device__ void initializeSpace(int N, float *waveSpace)
{
	int m = blockIdx.y * blockDim.y + threadIdx.y;
	int n = blockIdx.x * blockDim.x + threadIdx.x;

	//printf("m, n: %d, %d\n", m, n);

	/*for (int i = 0; i < blockDim.x; i++)
	{
		waveSpace[m * blockDim.x + i] = 0;
	}*/
	waveSpace[m * N + n] = 0;
	
	if ((m >= 0.4 * N) && (m < 0.6 * N) && (n >= 0.4 * N) && (n < 0.6 * N))
		//printf("m, n: %d, %d\n", m, n);
		waveSpace[m * N + n] = 20;

	/*for (int i = 0.4 * N; i < 0.6 * N; i++)
		for (int j = 0.4 * N; j < 0.6 * N; j++)
			waveSpace[N * i + j] = 20;*/
}

// Filling wave space with T > 1.
__device__ void fillSpaceTSteps(int N, int T, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1, float *waveSpaceTMin2)
{
	for (int i = 1; i < N; i++)
		for (int j = 1; j < N - 1; j++)
			waveSpace[N * i + j] = 2 * waveSpaceTMin1[N * i + j] - waveSpaceTMin2[N * i + j] + (c * c) * (dt/dd * dt/dd) * (waveSpaceTMin1[N * (i + 1) + j] + waveSpaceTMin1[N * (i - 1) + j] + waveSpaceTMin1[N * i + (j - 1)] + waveSpaceTMin1[N * i + (j + 1)] - 4 * waveSpaceTMin1[N * i + j]);
}

__global__ void schroedinger(float *waveSpace, float *waveSpaceTMin1, float *waveSpaceTMin2, int T, int N, char *f, int t)
{
	float c = 1.0;
	float dt = 0.1;
	float dd = 2.0;

	//printf("blockIdx: %d\n", blockIdx.y);

	// Schroedinger ecuation, by a given step as input.
	for (int step = 0; step <= T; step++)
	{
		//printf("step: %d\n", step);
		switch(step)
		{
			case 0:
				initializeSpace(N, waveSpace);
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
			case 1:
				fillSpaceFirstStep(N, c, dt, dd, waveSpace, waveSpaceTMin1);
				memcpy(waveSpaceTMin2, waveSpaceTMin1, N * N * sizeof(float));
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
			default:
				fillSpaceTSteps(N, T, c, dt, dd, waveSpace, waveSpaceTMin1, waveSpaceTMin2);
				memcpy(waveSpaceTMin2, waveSpaceTMin1, N * N * sizeof(float));
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
		}

		// Save step image specified by parameter t.
		if (step == t)
			break;
			//setWaveSpace(N, f, waveSpace);
	}
}

__host__ int main(int argc, char **argv)
{
	double start = omp_get_wtime();

	dim3 numblocks;
	dim3 sizeblocks;	
	
	int N, T, X, Y, t;
	
	int selection;
	char *f;
	
	while ((selection = getopt(argc, argv, "N:T:X:Y:f:t:")) != -1)
	{
		switch(selection) {
			case 'N':
				N = atoi(optarg);
				break;
			case 'T':
				T = atoi(optarg);
				break;
			case 'X':
				X = atoi(optarg);
				break;
			case 'Y':
				Y = atoi(optarg);
				break;
			case 'f':
				f = optarg;
				remove(f);
				break;
			case 't':
				t = atoi(optarg);
				break;
		}
	}
	
	if (t > T)
	{
		printf("Error: output iteration can't be bigger than step quantity.\n");
		exit(1);
	}

	numblocks.x = N/X;
	numblocks.y = N/Y;

	sizeblocks.x = X;
	sizeblocks.y = Y;

	//printf("numBlocks: (%d, %d)\n", numblocks.x, numblocks.y);
	//printf("sizeBlocks: (%d, %d)\n", sizeblocks.x, sizeblocks.y);

	// Setting wave spaces, saving states t, t - 1 and t - 2.
	float *waveSpace;
	waveSpace = (float *)malloc(N * N * sizeof(float));
	/*float *waveSpace, *waveSpaceTMin1, *waveSpaceTMin2;
	waveSpace = (float *)malloc(N * N * sizeof(float));
	waveSpaceTMin1 = (float *)malloc(N * N * sizeof(float));
	waveSpaceTMin2 = (float *)malloc(N * N * sizeof(float));*/
	
	// Setting wave spaces in GPU.
	float *waveSpace_d, *waveSpaceTMin1_d, *waveSpaceTMin2_d;
	hipMalloc((void**)&waveSpace_d, N * N * sizeof(float));
	hipMalloc((void**)&waveSpaceTMin1_d, N * N * sizeof(float));
	hipMalloc((void**)&waveSpaceTMin2_d, N * N * sizeof(float));

	// Copying values from CPU to GPU.
	/*cudaMemcpy(waveSpace_d, waveSpace_d, N * N * sizeof(float), cudaMemcpyHostToDevice);
	cudaMemcpy(waveSpaceTMin1_d, waveSpaceTMin1_d, N * N * sizeof(float), cudaMemcpyHostToDevice);
	cudaMemcpy(waveSpaceTMin2_d, waveSpaceTMin2_d, N * N * sizeof(float), cudaMemcpyHostToDevice);*/

	// Executing kernel.
	schroedinger<<<numblocks,sizeblocks>>>(waveSpace_d, waveSpaceTMin1_d, waveSpaceTMin2_d, T, N, f, t);
	hipDeviceSynchronize();
	
	hipMemcpy(waveSpace, waveSpace_d, N * N * sizeof(float), hipMemcpyDeviceToHost);

	setWaveSpace(N, f, waveSpace);

	double end = omp_get_wtime();
	
	printf("Time spent: %f\n", end - start);
	
	return 0;
}
