
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <getopt.h>
#include <string.h>
#include <omp.h>

// Save wave state as binary file.
int setWaveSpace(int N, char *f, float *waveSpace)
{
	FILE *filestream = fopen(f, "w+b");
	
	if (!filestream)
	{
		printf("Couldn't open file.\n");
		return 1;
	}
	
	for (int i = 0; i < N; i++)
	{
		fwrite(&waveSpace[i], sizeof(float), N * N, filestream);
		// Print wave space as CSV format.
		for (int j = 0; j < N; j++)
			printf("%f;", waveSpace[N * i + j]);
		printf("\n");
	}
	
	fclose(filestream);

	return 0;
}

// Read a wave state from binary file.
int getWaveSpace(int N, char *f)
{
	FILE *filestream = fopen(f, "r+b");
	float ws[N * N];

	if (!filestream)
	{
		return 1;
	}

	fread(&ws, sizeof(float), N * N, filestream);
	
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
			printf("%f;", ws[N * i + j]);
		printf("\n");
	}
	
	fclose(filestream);

	return 0;
}

// Filling wave space with T = 1.
void fillSpaceFirstStep(int N, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1)
{
	for (int i = 1; i < N; i++)
		for (int j = 1; j < N - 1; j++)
			waveSpace[N * i + j] = waveSpaceTMin1[N * i + j] + (c * c)/2 * (dt/dd * dt/dd) * (waveSpaceTMin1[N * (i + 1) + j] + waveSpaceTMin1[N * (i - 1) + j] + waveSpaceTMin1[N * i + (j - 1)] + waveSpaceTMin1[N * i + (j + 1)] - 4 * waveSpaceTMin1[N * i + j]);
}

// Setting wave space as a grid with values "20" in center
// sorrounding with zero values.
void initializeSpace(int N, float *waveSpace)
{
	for (int i = 0; i < N * N; i++)
		waveSpace[i] = 0;
	
	for (int i = 0.4 * N; i < 0.6 * N; i++)
		for (int j = 0.4 * N; j < 0.6 * N; j++)
			waveSpace[N * i + j] = 20;
}

// Filling wave space with T > 1.
__device__ void fillSpaceTSteps(int N, int T, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1, float *waveSpaceTMin2)
{

	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	/*for (int i = 1; i < N; i++)
		for (int j = 1; j < N - 1; j++)*/
			waveSpace[N * i + j] = 2 * waveSpaceTMin1[N * i + j] - waveSpaceTMin2[N * i + j] + (c * c) * (dt/dd * dt/dd) * (waveSpaceTMin1[N * (i + 1) + j] + waveSpaceTMin1[N * (i - 1) + j] + waveSpaceTMin1[N * i + (j - 1)] + waveSpaceTMin1[N * i + (j + 1)] - 4 * waveSpaceTMin1[N * i + j]);

	__syncthreads();
}

__global__ void schroedinger(float *waveSpace, float *waveSpaceTMin1, float *waveSpaceTMin2, int T, int N, char *f, int t)
{
	float c = 1.0;
	float dt = 0.1;
	float dd = 2.0;

	//printf("Inside kernel.\n");

	// Schroedinger ecuation, by a given step as input.
	for (int step = 2; step <= T; step++)
	{
		//printf("step: %d\n", step);
		/*switch(step)
		{
			case 0:
				initializeSpace(N, waveSpace);
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
			case 1:
				fillSpaceFirstStep(N, c, dt, dd, waveSpace, waveSpaceTMin1);
				memcpy(waveSpaceTMin2, waveSpaceTMin1, N * N * sizeof(float));
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
			default:*/
				fillSpaceTSteps(N, T, c, dt, dd, waveSpace, waveSpaceTMin1, waveSpaceTMin2);
				memcpy(waveSpaceTMin2, waveSpaceTMin1, N * N * sizeof(float));
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				/*break;
		}*/

		// Save step image specified by parameter t.
		//if (step == t)
		//	break;
			//setWaveSpace(N, f, waveSpace);
	}
}

__host__ int main(int argc, char **argv)
{
	double start = omp_get_wtime();

	dim3 numblocks;
	dim3 sizeblocks;	
	
	int N, T, X, Y, t;
	
	int selection;
	char *f;
	
	while ((selection = getopt(argc, argv, "N:T:X:Y:f:t:")) != -1)
	{
		switch(selection) {
			case 'N':
				N = atoi(optarg);
				break;
			case 'T':
				T = atoi(optarg);
				break;
			case 'X':
				X = atoi(optarg);
				break;
			case 'Y':
				Y = atoi(optarg);
				break;
			case 'f':
				f = optarg;
				remove(f);
				break;
			case 't':
				t = atoi(optarg);
				break;
		}
	}
	
	if (t > T)
	{
		printf("Error: output iteration can't be bigger than step quantity.\n");
		exit(1);
	}

	numblocks.x = (int)ceil((float)N/X);
	numblocks.y = (int)ceil((float)N/Y);

	/*printf("N: %d\n", N);
	printf("Y: %d\n", Y);
	printf("numblocks.x: %d\n", numblocks.x);
	printf("ceil: %d\n", (int)ceil((float)N/X));*/

	sizeblocks.x = X;
	sizeblocks.y = Y;

	//printf("numBlocks: (%d, %d)\n", numblocks.x, numblocks.y);
	//printf("sizeBlocks: (%d, %d)\n", sizeblocks.x, sizeblocks.y);

	// Setting wave spaces, saving states t, t - 1 and t - 2.
	float *waveSpace, *waveSpaceTMin1, *waveSpaceTMin2;
	waveSpace = (float *)malloc(N * N * sizeof(float));
	waveSpaceTMin1 = (float *)malloc(N * N * sizeof(float));
	waveSpaceTMin2 = (float *)malloc(N * N * sizeof(float));
	
	// Setting wave spaces in GPU.
	float *waveSpace_d, *waveSpaceTMin1_d, *waveSpaceTMin2_d;
	hipMalloc((void**)&waveSpace_d, N * N * sizeof(float));
	hipMalloc((void**)&waveSpaceTMin1_d, N * N * sizeof(float));
	hipMalloc((void**)&waveSpaceTMin2_d, N * N * sizeof(float));

	float c = 1.0;
	float dt = 0.1;
	float dd = 2.0;
	//printf("T: %d\n", T);
	for (int step = 0; step <= T; step++)
	{
		//printf("step: %d\n", step);
		switch(step)
		{
			case 0:
				initializeSpace(N, waveSpace);
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
			case 1:
				fillSpaceFirstStep(N, c, dt, dd, waveSpace, waveSpaceTMin1);
				memcpy(waveSpaceTMin2, waveSpaceTMin1, N * N * sizeof(float));
				memcpy(waveSpaceTMin1, waveSpace, N * N * sizeof(float));
				break;
			default:
				//printf("default\n");
				break;
		}
		
		if (step == 2)
		{
			//printf("step 2\n");
			break;
		}
	}

	// Copying values from CPU to GPU.
	hipMemcpy(waveSpace_d, waveSpace, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(waveSpaceTMin1_d, waveSpaceTMin1, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(waveSpaceTMin2_d, waveSpaceTMin2, N * N * sizeof(float), hipMemcpyHostToDevice);
	//printf("executing kernel...\n");
	// Executing kernel.
	schroedinger<<<numblocks,sizeblocks>>>(waveSpace_d, waveSpaceTMin1_d, waveSpaceTMin2_d, T, N, f, t);
	hipDeviceSynchronize();
	
	if (T >= 2)
	{
		printf("T: %d\n", T);
		hipMemcpy(waveSpace, waveSpace_d, N * N * sizeof(float), hipMemcpyDeviceToHost);
	}

	setWaveSpace(N, f, waveSpace);

	double end = omp_get_wtime();
	
	printf("Time spent: %f\n", end - start);
	
	return 0;
}
